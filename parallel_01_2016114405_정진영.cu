#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> // for rand(), malloc(), free()
#include <windows.h> // for high-resolution performance counter

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do {\
		(x); \
		hipError_t e = hipGetLastError(); \
		if (hipSuccess != e) { \
			printf("cuda failure \"%s\" at %s:%d\n", \
				hipGetErrorString(e), \
			     __FILE__, __LINE__); \
			exit(1); \
		} \
	} while (0)
#endif

#define WIDTH		(4 * 1024)	// total width 4K
#define	TILE_WIDTH	32		// block will be (TILE_WIDTH,TILE_WIDTH)
#define	GRID_WIDTH	(WIDTH / TILE_WIDTH)	// grid will be (GRID_WDITH,GRID_WDITH)


void genData(float* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (float)(rand() % 1000) / 1000.0F;
	}
}


__global__ void matmul(float* c, const float* a, const float* b, const int width) {
	// c[y][x] = sum_k a[y][k] * b[k][x]
	// c[y * WIDTH + x] = sum_k a[y*WIDTH + k] * b[k*WIDTH + x]
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0F;
	for (register int k = 0; k < width; ++k) {
		float lhs = a[y * width + k];
		float rhs = b[k * width + x];
		sum += lhs * rhs;
	}
	c[y * width + x] = sum;
}


int main(void) {
	float* pM = NULL;
	float* pN = NULL;
	float* pP = NULL;
	long long cntStart, cntEnd, freq;
	QueryPerformanceFrequency((LARGE_INTEGER*)(&freq));
	// malloc memories on the host-side
	pM = (float*)malloc(WIDTH * WIDTH * sizeof(float));
	pN = (float*)malloc(WIDTH * WIDTH * sizeof(float));
	pP = (float*)malloc(WIDTH * WIDTH * sizeof(float));
	printf("pM, pN, pP = %#x %#x %#x\n", pM, pN, pP);
		// generate source data
	genData(pM, WIDTH * WIDTH);
	genData(pN, WIDTH * WIDTH);
	// CUDA: allocate device memory
	float* pMdev = NULL;
	float* pNdev = NULL;
	float* pPdev = NULL;
	CUDA_CHECK(hipMalloc((void**)&pMdev, WIDTH * WIDTH * sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&pNdev, WIDTH * WIDTH * sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&pPdev, WIDTH * WIDTH * sizeof(float)));
	printf("pMdev, pNdev, pPdev = %#x %#x %#x\n", pMdev, pNdev, pPdev);
		// CUDA: copy from host to device
	CUDA_CHECK(hipMemcpy(pMdev, pM, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(pNdev, pN, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice));
	// start the timer
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart)); // start the stop watch
	// CUDA: launch the kernel
	dim3 dimGrid(GRID_WIDTH, GRID_WIDTH, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	matmul << < dimGrid, dimBlock >> > (pPdev, pMdev, pNdev, WIDTH);
	// end the timer
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd)); // end the stop watch
	CUDA_CHECK(hipPeekAtLastError());
	printf("elapsed time = %f msec\n", (double)(cntEnd - cntStart) * 1000.0 / (double)(freq));
	// CUDA: copy from device to host
	CUDA_CHECK(hipMemcpy(pP, pPdev, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost));
	// print sample cases
	int i, j;
	i = 0; j = 0; printf("c[%4d][%4d] = %f\n", i, j, pP[i * WIDTH + j]);
	i = WIDTH / 2; j = WIDTH / 2; printf("c[%4d][%4d] = %f\n", i, j, pP[i * WIDTH + j]);
	i = WIDTH - 1; j = WIDTH - 1; printf("c[%4d][%4d] = %f\n", i, j, pP[i * WIDTH + j]);
	// CUDA: free the memory
	CUDA_CHECK(hipFree(pMdev));
	CUDA_CHECK(hipFree(pNdev));
	CUDA_CHECK(hipFree(pPdev));
	// free the memory
	free(pM);
	free(pN);
	free(pP);
}

